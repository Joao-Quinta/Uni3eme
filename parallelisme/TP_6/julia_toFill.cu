#include "hip/hip_runtime.h"
#include "stdio.h"
#include <iostream>
#include "Array2D.hpp"
#include <fstream>
#include <chrono>


__device__ double2 cplxmult(double2 a, double2 b){
    return make_double2(a.x*b.x-a.y*b.y, a.x*b.y+a.y*b.x);
}

__device__ double2 cplxadd(double2 a, double2 b){
    return make_double2(a.x+b.x, a.y+b.y);
}

__device__ double norm(double2 a){
    return sqrt(a.x*a.x + a.y*a.y);
}

__device__ int divergence(double2 z0, double2 c, double bound, int imax){
    double2 z = make_double2(z0.x,z0.y);
    for(int i = 0; i < imax; i++){
      if(norm(z) > bound) return i;
      z = cplxadd(cplxmult(z,z),c);
    }
    return imax;
}

__device__ double2 coord2cplx(double2 ll, double2 ur, int2 pos, int2 size){
  double2 res = make_double2(ll.x + pos.x*(ur.x - ll.x)/size.x, -(ll.y + pos.y*(ur.y - ll.y)/size.y);
    return res;
}

// Device code
__global__ void julia(int* A, int2 size, double2 ll, double2 ur, double2 c, double bound, int imax )
{
    for( int y = 0; y < size.y; y++){
      for( int x = 0; x < size.x; x++){
        int2 pos = make_int2(x,y);
        A[pos.y*size.x+pos.x] = divergence(coord2cplx(ll, ur, pos, size), c, bound, imax);
      }
    }
}

void writePgm(Array2D<int>& d, int imax, std::string filename){
    std::ofstream file;
    file.open (filename);
    file << "P2" << std::endl;
    file << d.sizeX() << " " << d.sizeY() << std::endl;
    file << imax << std::endl;
    for(int y=0; y<d.sizeY(); y++){
        for(int x=0; x<d.sizeX(); x++){
            file << d(x, y) << " ";
        }
        file << std::endl;
    }
    file.close();
}

// Host code
int main(int argc, char** argv)
{
    int2 size = make_int2(std::atoi(argv[1]), std::atoi(argv[2]));
    double2 ll = make_double2(std::atof(argv[3]), std::atof(argv[4]));
    double2 ur = make_double2(std::atof(argv[5]), std::atof(argv[6]));
    double2 c = make_double2(std::atof(argv[7]), std::atof(argv[8]));
    double bound = std::atof(argv[9]);
    int imax = std::atoi(argv[10]);

    // Allocate vectors h_A, h_B and h_C in host memory
    Array2D<int> h_A(size.x, size.y);

    // Allocate vectors in device memory
    int* d_A;
    hipMalloc(&d_A, size.x*size.y*sizeof(int));

    // Invoke kernel
    dim3 dimBlock(16,16);
    dim3 dimGrid((size.x + dimBlock.x - 1)/dimBlock.x, (size.y + dimBlock.y - 1)/dimBlock.y);


    auto start = std::chrono::steady_clock::now();

    julia<<<dimGrid, dimBlock>>>(d_A, size, ll, ur, c, bound, imax);

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();

    auto diff = end - start;
    std::cout << "Computation time : " << std::chrono::duration <double, std::milli> (diff).count() << " ms" << std::endl;

    // Copy result from device memory to host memory
    // h_C contains the result in host memory

    start = std::chrono::steady_clock::now();

    hipMemcpy(h_A.data(), d_A, size.x*size.y*sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();
    diff = end - start;
    std::cout << "Copy time : " << std::chrono::duration <double, std::milli> (diff).count() << " ms" << std::endl;

    writePgm(h_A, imax, "julia.pgm");

    // Free device memory
    hipFree(d_A);
}
