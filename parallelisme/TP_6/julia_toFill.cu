#include "hip/hip_runtime.h"
#include "stdio.h"
#include <iostream>
#include "Array2D.hpp"
#include <fstream>
#include <chrono>


__device__ double2 cplxmult(double2 a, double2 b){
    return make_double2(a.x*b.x-a.y*b.y, a.x*b.y+a.y*b.x);
}

__device__ double2 cplxadd(double2 a, double2 b){
    return make_double2(a.x+b.x, a.y+b.y);
}

__device__ double norm(double2 a){
    return sqrt(a.x*a.x + a.y*a.y);
}

__...__ int divergence(double2 z0, double2 c, double bound, int imax){
    ...
}

__...__ double2 coord2cplx(double2 ll, double2 ur, int2 pos, int2 size){
    return ...;
}

// Device code
__...__ void julia(int* A, int2 size, double2 ll, double2 ur, double2 c, double bound, int imax )
{
    ...
}

void writePgm(Array2D<int>& d, int imax, std::string filename){
    std::ofstream file;
    file.open (filename);
    file << "P2" << std::endl;
    file << d.sizeX() << " " << d.sizeY() << std::endl;
    file << imax << std::endl;
    for(int y=0; y<d.sizeY(); y++){
        for(int x=0; x<d.sizeX(); x++){
            file << d(x, y) << " ";
        }
        file << std::endl;
    }
    file.close();
}

// Host code
int main(int argc, char** argv)
{
    int2 size = make_int2(std::atoi(argv[1]), std::atoi(argv[2]));
    double2 ll = make_double2(std::atof(argv[3]), std::atof(argv[4]));
    double2 ur = make_double2(std::atof(argv[5]), std::atof(argv[6]));
    double2 c = make_double2(std::atof(argv[7]), std::atof(argv[8]));
    double bound = std::atof(argv[9]);
    int imax = std::atoi(argv[10]);

    // Allocate vectors h_A, h_B and h_C in host memory
    Array2D<int> h_A(size.x, size.y);

    // Allocate vectors in device memory
    int* d_A;
    hipMalloc(...);

    // Invoke kernel
    dim3 dimBlock(...);
    dim3 dimGrid(...);


    auto start = std::chrono::steady_clock::now();

    julia<<<dimGrid, dimBlock>>>(d_A, size, ll, ur, c, bound, imax);

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();

    auto diff = end - start;
    std::cout << "Computation time : " << std::chrono::duration <double, std::milli> (diff).count() << " ms" << std::endl;

    // Copy result from device memory to host memory
    // h_C contains the result in host memory

    start = std::chrono::steady_clock::now();

    hipMemcpy(...);

    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();
    diff = end - start;
    std::cout << "Copy time : " << std::chrono::duration <double, std::milli> (diff).count() << " ms" << std::endl;

    writePgm(h_A, imax, "julia.pgm");

    // Free device memory
    hipFree(...);
}
